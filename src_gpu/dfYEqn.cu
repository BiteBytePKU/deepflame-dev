#include "hip/hip_runtime.h"
#include "dfYEqn.H"

// kernel functions
__global__ void getUpwindWeight(int num_faces, double *phi, double *weight)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_faces)
        return;
    if (phi[index] >= 0)
        weight[index] = 1.;
    else
        weight[index] = 0.;
}

__global__ void fvc_grad_internal(int num_cells, int num_species,
        const int *csr_row_index, const int *csr_col_index, const int *csr_diag_index,
        const double *face_vector, const double *weight, const double *species,
        const double *volume, double *grady)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;

    // A_csr has one more element in each row: itself
    int row_index = csr_row_index[index];
    int next_row_index = csr_row_index[index + 1];
    int diag_index = csr_diag_index[index];
    int neighbor_offset = csr_row_index[index] - index;

    double vol = volume[index];

    for (int s = 0; s < num_species; s++) {
    double own_cell_Y = species[num_cells * s + index];
    double grad_bx = 0;
    double grad_by = 0;
    double grad_bz = 0;
    for (int i = row_index; i < next_row_index; i++)
    {
        int inner_index = i - row_index;
        // lower
        if (inner_index < diag_index)
        {
            int neighbor_index = neighbor_offset + inner_index;
            double w = weight[neighbor_index];
            double sfx = face_vector[neighbor_index * 3 + 0];
            double sfy = face_vector[neighbor_index * 3 + 1];
            double sfz = face_vector[neighbor_index * 3 + 2];
            int neighbor_cell_id = csr_col_index[row_index + inner_index];
            double neighbor_cell_Y = species[num_cells * s + neighbor_cell_id];
            double face_Y = w * (neighbor_cell_Y - own_cell_Y) + own_cell_Y;
            grad_bx -= face_Y * sfx;
            grad_by -= face_Y * sfy;
            grad_bz -= face_Y * sfz;
        }
        // upper
        if (inner_index > diag_index)
        {
            int neighbor_index = neighbor_offset + inner_index - 1;
            double w = weight[neighbor_index];
            double sfx = face_vector[neighbor_index * 3 + 0];
            double sfy = face_vector[neighbor_index * 3 + 1];
            double sfz = face_vector[neighbor_index * 3 + 2];
            int neighbor_cell_id = csr_col_index[row_index + inner_index];
            double neighbor_cell_Y = species[num_cells * s + neighbor_cell_id];
            double face_Y = w * (own_cell_Y - neighbor_cell_Y) + neighbor_cell_Y;
            grad_bx += face_Y * sfx;
            grad_by += face_Y * sfy;
            grad_bz += face_Y * sfz;
        }
    }
    grady[num_cells * s * 3 + index * 3 + 0] = grad_bx / vol;
    grady[num_cells * s * 3 + index * 3 + 1] = grad_by / vol;
    grady[num_cells * s * 3 + index * 3 + 2] = grad_bz / vol;
    }
}
__global__ void fvc_grad_boundary(int num_cells, int num_boundary_cells, int num_boundary_faces, int num_species,
        const int *boundary_cell_offset, const int *boundary_cell_id, const int *bouPermedIndex,
        const double *boundary_face_vector, const double *boundary_species_init,
        const double *volume, const double *grady_input, double *grady_output, bool uploadBoundaryY)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_boundary_cells)
        return;

    int cell_offset = boundary_cell_offset[index];
    int next_cell_offset = boundary_cell_offset[index + 1];
    int cell_index = boundary_cell_id[cell_offset];

    double vol = volume[index];

    // compute boundary gradient
    for (int s = 0; s < num_species; s++) {
    double grad_bx = 0;
    double grad_by = 0;
    double grad_bz = 0;
    for (int i = cell_offset; i < next_cell_offset; i++)
    {
        double sfx = boundary_face_vector[i * 3 + 0];
        double sfy = boundary_face_vector[i * 3 + 1];
        double sfz = boundary_face_vector[i * 3 + 2];
        double face_Y;
        if (!uploadBoundaryY)
        {
            face_Y = boundary_species_init[num_boundary_faces * s + i];
        }
        else
        {
            int permute_index = bouPermedIndex[i];
            face_Y = boundary_species_init[num_boundary_faces * s + permute_index];
        }
        grad_bx += face_Y * sfx;
        grad_by += face_Y * sfy;
        grad_bz += face_Y * sfz;
    }

    grady_output[num_cells * s * 3 + cell_index * 3 + 0] =
        grady_input[num_cells * s * 3 + cell_index * 3 + 0] + grad_bx / vol;
    grady_output[num_cells * s * 3 + cell_index * 3 + 1] =
        grady_input[num_cells * s * 3 + cell_index * 3 + 1] + grad_by / vol;
    grady_output[num_cells * s * 3 + cell_index * 3 + 2] =
        grady_input[num_cells * s * 3 + cell_index * 3 + 2] + grad_bz / vol;
    }
}
__global__ void correct_boundary_conditions(int num_cells, int num_boundary_cells, int num_boundary_faces, int num_species,
                                                const int *boundary_cell_offset, const int *boundary_cell_id,
                                                const double *boundary_sf, const double *mag_sf,
                                                const double *grady, double* boundary_grady)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_boundary_cells)
        return;

    int cell_offset = boundary_cell_offset[index];
    int next_cell_offset = boundary_cell_offset[index + 1];
    int cell_index = boundary_cell_id[cell_offset];

    for (int s = 0; s < num_species; s++) {
    // initialize boundary_sumYDiffError
    double grady_x = grady[num_cells * s * 3 + cell_index * 3 + 0];
    double grady_y = grady[num_cells * s * 3 + cell_index * 3 + 1];
    double grady_z = grady[num_cells * s * 3 + cell_index * 3 + 2];

    for (int i = cell_offset; i < next_cell_offset; i++)
    {
        double n_x = boundary_sf[i * 3 + 0] / mag_sf[i];
        double n_y = boundary_sf[i * 3 + 1] / mag_sf[i];
        double n_z = boundary_sf[i * 3 + 2] / mag_sf[i];
        double sn_grad = 0;
        double grad_correction = sn_grad - (n_x * grady_x + n_y * grady_y + n_z * grady_z);
        boundary_grady[num_boundary_faces * s * 3 + i * 3 + 0] = grady_x + grad_correction * n_x;
        boundary_grady[num_boundary_faces * s * 3 + i * 3 + 1] = grady_y + grad_correction * n_y;
        boundary_grady[num_boundary_faces * s * 3 + i * 3 + 2] = grady_z + grad_correction * n_z;
    }
    }
}

__global__ void sumError_internal(int num_cells, int num_species,
        const double *hai, const double *rhoD, const double *y, const double *grady,
        double *sum_hai_rhoD_grady, double *sum_rhoD_grady, double *sum_hai_y)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;

    double sum_hai_rhoD_grady_x = 0;
    double sum_hai_rhoD_grady_y = 0;
    double sum_hai_rhoD_grady_z = 0;
    double sum_rhoD_grady_x = 0;
    double sum_rhoD_grady_y = 0;
    double sum_rhoD_grady_z = 0;
    double sum_hai_y_value = 0;
    for (int s = 0; s < num_species; s++) {
        double hai_value = hai[num_cells * s + index];
        double rhoD_value = rhoD[num_cells * s + index];
        double y_value = y[num_cells * s + index];
        double grady_x = grady[num_cells * s * 3 + index * 3 + 0];
        double grady_y = grady[num_cells * s * 3 + index * 3 + 1];
        double grady_z = grady[num_cells * s * 3 + index * 3 + 2];
        sum_hai_rhoD_grady_x += hai_value * rhoD_value * grady_x;
        sum_hai_rhoD_grady_y += hai_value * rhoD_value * grady_y;
        sum_hai_rhoD_grady_z += hai_value * rhoD_value * grady_z;
        sum_rhoD_grady_x += rhoD_value * grady_x;
        sum_rhoD_grady_y += rhoD_value * grady_y;
        sum_rhoD_grady_z += rhoD_value * grady_z;
        sum_hai_y_value += hai_value * y_value;
    }
    sum_hai_rhoD_grady[index * 3 + 0] = sum_hai_rhoD_grady_x;
    sum_hai_rhoD_grady[index * 3 + 1] = sum_hai_rhoD_grady_y;
    sum_hai_rhoD_grady[index * 3 + 2] = sum_hai_rhoD_grady_z;
    sum_rhoD_grady[index * 3 + 0] = sum_rhoD_grady_x;
    sum_rhoD_grady[index * 3 + 1] = sum_rhoD_grady_y;
    sum_rhoD_grady[index * 3 + 2] = sum_rhoD_grady_z;
    sum_hai_y[index] = sum_hai_y_value;
}

__global__ void sumError_boundary(int num_boundary_faces, int num_species, const int *bouPermedIndex,
        const double *boundary_hai, const double *boundary_rhoD, const double *boundary_y, const double *boundary_grady,
        double *sum_boundary_hai_rhoD_grady, double *sum_boundary_rhoD_grady, double *sum_boundary_hai_y, bool uploadBoundaryY)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_boundary_faces)
        return;

    int permute_index;
    if (!uploadBoundaryY)
    {
        permute_index = index;
    }
    else
    {
        permute_index = bouPermedIndex[index];
    }
    double sum_boundary_hai_rhoD_grady_x = 0;
    double sum_boundary_hai_rhoD_grady_y = 0;
    double sum_boundary_hai_rhoD_grady_z = 0;
    double sum_boundary_rhoD_grady_x = 0;
    double sum_boundary_rhoD_grady_y = 0;
    double sum_boundary_rhoD_grady_z = 0;
    double sum_boundary_hai_y_value = 0;
    for (int s = 0; s < num_species; s++) {
        double boundary_hai_value = boundary_hai[num_boundary_faces * s + permute_index];
        double boundary_rhoD_value = boundary_rhoD[num_boundary_faces * s + permute_index];
        double boundary_y_value = boundary_y[num_boundary_faces * s + permute_index];
        double boundary_grady_x = boundary_grady[num_boundary_faces * s * 3 + index * 3 + 0];
        double boundary_grady_y = boundary_grady[num_boundary_faces * s * 3 + index * 3 + 1];
        double boundary_grady_z = boundary_grady[num_boundary_faces * s * 3 + index * 3 + 2];
        sum_boundary_hai_rhoD_grady_x += boundary_hai_value * boundary_rhoD_value * boundary_grady_x;
        sum_boundary_hai_rhoD_grady_y += boundary_hai_value * boundary_rhoD_value * boundary_grady_y;
        sum_boundary_hai_rhoD_grady_z += boundary_hai_value * boundary_rhoD_value * boundary_grady_z;
        sum_boundary_rhoD_grady_x += boundary_rhoD_value * boundary_grady_x;
        sum_boundary_rhoD_grady_y += boundary_rhoD_value * boundary_grady_y;
        sum_boundary_rhoD_grady_z += boundary_rhoD_value * boundary_grady_z;
        sum_boundary_hai_y_value += boundary_hai_value * boundary_y_value;
    }
    sum_boundary_hai_rhoD_grady[index * 3 + 0] = sum_boundary_hai_rhoD_grady_x;
    sum_boundary_hai_rhoD_grady[index * 3 + 1] = sum_boundary_hai_rhoD_grady_y;
    sum_boundary_hai_rhoD_grady[index * 3 + 2] = sum_boundary_hai_rhoD_grady_z;
    sum_boundary_rhoD_grady[index * 3 + 0] = sum_boundary_rhoD_grady_x;
    sum_boundary_rhoD_grady[index * 3 + 1] = sum_boundary_rhoD_grady_y;
    sum_boundary_rhoD_grady[index * 3 + 2] = sum_boundary_rhoD_grady_z;
    sum_boundary_hai_y[index] = sum_boundary_hai_y_value;
}

__global__ void calculate_hDiffCorrFlux(int num,
        const double *sum_hai_rhoD_grady, const double *sum_rhoD_grady, const double *sum_hai_y, double *hDiffCorrFlux)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;

    hDiffCorrFlux[index * 3 + 0] += (sum_hai_rhoD_grady[index * 3 + 0] - sum_hai_y[index] * sum_rhoD_grady[index * 3 + 0]);
    hDiffCorrFlux[index * 3 + 1] += (sum_hai_rhoD_grady[index * 3 + 1] - sum_hai_y[index] * sum_rhoD_grady[index * 3 + 1]);
    hDiffCorrFlux[index * 3 + 2] += (sum_hai_rhoD_grady[index * 3 + 2] - sum_hai_y[index] * sum_rhoD_grady[index * 3 + 2]);
}

__global__ void calculate_phiUc_internal(int num_cells,
        const int *csr_row_index, const int *csr_col_index, const int *csr_diag_index,
        const double *face_vector, const double *weight, const double *sumYDiffError, double *phiUc)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;

    // A_csr has one more element in each row: itself
    int row_index = csr_row_index[index];
    int row_elements = csr_row_index[index + 1] - row_index;
    int diag_index = csr_diag_index[index];
    int neighbor_offset = csr_row_index[index] - index;

    double own_cell_sumYDiffError_x = sumYDiffError[index * 3 + 0];
    double own_cell_sumYDiffError_y = sumYDiffError[index * 3 + 1];
    double own_cell_sumYDiffError_z = sumYDiffError[index * 3 + 2];

    // lower
    for (int i = 0; i < diag_index; i++)
    {
        double phiUc_face = 0;

        int neighbor_index = neighbor_offset + i;
        int neighbor_cell_id = csr_col_index[row_index + i];
        double w = weight[neighbor_index];
        double sfx = face_vector[neighbor_index * 3 + 0];
        double sfy = face_vector[neighbor_index * 3 + 1];
        double sfz = face_vector[neighbor_index * 3 + 2];
        double neighbor_cell_sumYDiffError_x = sumYDiffError[neighbor_cell_id * 3 + 0];
        double neighbor_cell_sumYDiffError_y = sumYDiffError[neighbor_cell_id * 3 + 1];
        double neighbor_cell_sumYDiffError_z = sumYDiffError[neighbor_cell_id * 3 + 2];
        double face_x = w * (neighbor_cell_sumYDiffError_x - own_cell_sumYDiffError_x) + own_cell_sumYDiffError_x;
        double face_y = w * (neighbor_cell_sumYDiffError_y - own_cell_sumYDiffError_y) + own_cell_sumYDiffError_y;
        double face_z = w * (neighbor_cell_sumYDiffError_z - own_cell_sumYDiffError_z) + own_cell_sumYDiffError_z;

        phiUc_face = face_x * sfx + face_y * sfy + face_z * sfz;
        phiUc[neighbor_index] = phiUc_face;
    }
    // upper
    for (int i = diag_index + 1; i < row_elements; i++)
    {
        double phiUc_face = 0;

        int neighbor_index = neighbor_offset + i - 1;
        int neighbor_cell_id = csr_col_index[row_index + i];
        double w = weight[neighbor_index];
        double sfx = face_vector[neighbor_index * 3 + 0];
        double sfy = face_vector[neighbor_index * 3 + 1];
        double sfz = face_vector[neighbor_index * 3 + 2];
        double neighbor_cell_sumYDiffError_x = sumYDiffError[neighbor_cell_id * 3 + 0];
        double neighbor_cell_sumYDiffError_y = sumYDiffError[neighbor_cell_id * 3 + 1];
        double neighbor_cell_sumYDiffError_z = sumYDiffError[neighbor_cell_id * 3 + 2];
        double face_x = w * (own_cell_sumYDiffError_x - neighbor_cell_sumYDiffError_x) + neighbor_cell_sumYDiffError_x;
        double face_y = w * (own_cell_sumYDiffError_y - neighbor_cell_sumYDiffError_y) + neighbor_cell_sumYDiffError_y;
        double face_z = w * (own_cell_sumYDiffError_z - neighbor_cell_sumYDiffError_z) + neighbor_cell_sumYDiffError_z;

        phiUc_face = face_x * sfx + face_y * sfy + face_z * sfz;
        phiUc[neighbor_index] = phiUc_face;
    }
}

__global__ void calculate_phiUc_boundary(int num_boundary_faces,
                                         const int *boundary_cell_offset, const int *boundary_cell_id,
                                         const double *boundary_sf, const double *boundary_sumYDiffError,
                                         double *boundary_phiUc)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_boundary_faces)
        return;

    double n_x = boundary_sf[index * 3 + 0];
    double n_y = boundary_sf[index * 3 + 1];
    double n_z = boundary_sf[index * 3 + 2];

    double err_x = boundary_sumYDiffError[index * 3 + 0];
    double err_y = boundary_sumYDiffError[index * 3 + 1];
    double err_z = boundary_sumYDiffError[index * 3 + 2];

    boundary_phiUc[index] = n_x * err_x + n_y * err_y + n_z * err_z;
}

__global__ void fvm_ddt_kernel_scalar(int num_cells, int num_faces, int num_species, int inertIndex, const double rdelta_t,
                                      const int *csr_row_index, const int *csr_diag_index,
                                      const double *rho_old, const double *rho_new, const double *volume, const double *species_old,
                                      const double *A_csr_input, const double *b_input, double *A_csr_output, double *b_output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;

    // A_csr has one more element in each row: itself
    int row_index = csr_row_index[index];
    int diag_index = csr_diag_index[index];
    int csr_index = row_index + diag_index;

    double ddt_diag = rdelta_t * rho_new[index] * volume[index];
    double ddt_part_term = rdelta_t * rho_old[index] * volume[index];
    int mtxIndex = 0;
    for (int s = 0; s < num_species; s++) {
        if (s == inertIndex)
            continue;
        A_csr_output[mtxIndex * (num_cells + num_faces) + csr_index] =
            A_csr_input[mtxIndex * (num_cells + num_faces) + csr_index] + ddt_diag;
        b_output[mtxIndex * num_cells + index] =
            b_input[mtxIndex * num_cells + index] + ddt_part_term * species_old[num_cells * s + index];
        ++mtxIndex;
    }
}

__global__ void compute_inertIndex_y(int num_cells, int num_species, int inertIndex, double *y)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;

    double sum_yi = 0;
    for (int i = 0; i < num_species; i++)
    {
        if (i == inertIndex) continue;

        double yi = y[num_cells * i + index];
        sum_yi += yi > 0 ? yi : 0;
    }
    sum_yi = 1 - sum_yi;
    y[num_cells * inertIndex + index] = (sum_yi > 0 ? sum_yi : 0);
}

__global__ void fvm_div_internal_scalar(int num_cells, int num_faces, int num_species, int inertIndex,
                                        const int *csr_row_index, const int *csr_diag_index,
                                        const double *div_weight, const double *phi,
                                        const double *A_csr_input, double *A_csr_output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;

    // A_csr has one more element in each row: itself
    int row_index = csr_row_index[index];
    int next_row_index = csr_row_index[index + 1];
    int diag_index = csr_diag_index[index];
    int neighbor_offset = csr_row_index[index] - index;

    int mtxIndex = 0;
    for (int s = 0; s < num_species; s++) {
        if (s == inertIndex)
            continue;
    double div_diag = 0;
    for (int i = row_index; i < next_row_index; i++)
    {
        int inner_index = i - row_index;
        // lower
        if (inner_index < diag_index)
        {
            int neighbor_index = neighbor_offset + inner_index;
            double w = div_weight[neighbor_index];
            double f = phi[neighbor_index];
            A_csr_output[mtxIndex * (num_cells + num_faces) + i] =
                A_csr_input[mtxIndex * (num_cells + num_faces) + i] + (-w) * f;
            // lower neighbors contribute to sum of -1
            div_diag += (w - 1) * f;
        }
        // upper
        if (inner_index > diag_index)
        {
            // upper, index - 1, consider of diag
            int neighbor_index = neighbor_offset + inner_index - 1;
            double w = div_weight[neighbor_index];
            double f = phi[neighbor_index];
            A_csr_output[mtxIndex * (num_cells + num_faces) + i] =
                A_csr_input[mtxIndex * (num_cells + num_faces) + i] + (1 - w) * f;
            // upper neighbors contribute to sum of 1
            div_diag += w * f;
        }
    }
    A_csr_output[mtxIndex * (num_cells + num_faces) + row_index + diag_index] =
        A_csr_input[mtxIndex * (num_cells + num_faces) + row_index + diag_index] + div_diag; // diag
        ++mtxIndex;
    }
}
__global__ void fvm_div_boundary_scalar(int num_cells, int num_faces, int num_boundary_cells, int num_boundary_faces,
                                        int num_species, int inertIndex,
                                        const int *csr_row_index, const int *csr_diag_index, const double *boundary_phi,
                                        const int *boundary_cell_offset, const int *boundary_cell_id,
                                        double *internal_coeffs, const double *boundary_coeffs,
                                        const double *A_csr_input, double *A_csr_output, const double *b_input, double *b_output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_boundary_cells)
        return;

    int cell_offset = boundary_cell_offset[index];
    int cell_index = boundary_cell_id[cell_offset];
    int loop_size = boundary_cell_offset[index + 1] - cell_offset;

    int row_index = csr_row_index[cell_index];
    int diag_index = csr_diag_index[cell_index];
    int csr_dim = num_cells + num_faces;
    int csr_index = row_index + diag_index;

    int mtxIndex = 0;
    for (int s = 0; s < num_species; s++) {
        if (s == inertIndex)
            continue;
    // construct internalCoeffs & boundaryCoeffs
    double internal_coeffs_own = 0;
    double boundary_coeffs_own = 0;
    for (int i = 0; i < loop_size; i++)
    {
        internal_coeffs_own += boundary_phi[cell_offset + i] * internal_coeffs[num_boundary_faces * s + cell_offset + i];
        boundary_coeffs_own += -boundary_phi[cell_offset + i] * boundary_coeffs[num_boundary_faces + s + cell_offset + i];
    }
    A_csr_output[mtxIndex * (num_cells + num_faces) + csr_index] =
        A_csr_input[mtxIndex * (num_cells + num_faces) + csr_index] + internal_coeffs_own;
    b_output[mtxIndex * num_cells + cell_index] =
        b_input[mtxIndex * num_cells + cell_index] + boundary_coeffs_own;
        ++mtxIndex;
    }
}

__global__ void fvm_laplacian_uncorrected_scalar_internal(int num_cells, int num_faces, int num_species, int inertIndex,
                                                          const int *csr_row_index, const int *csr_col_index, const int *csr_diag_index,
                                                          const double *mut_sct, const double *rhoD, const double *weight,
                                                          const double *magsf, const double *distance,
                                                          const double sign, const double *A_csr_input, double *A_csr_output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;

    // A_csr has one more element in each row: itself
    int row_index = csr_row_index[index];
    int row_elements = csr_row_index[index + 1] - row_index;
    int diag_index = csr_diag_index[index];
    int neighbor_offset = csr_row_index[index] - index;

    int mtxIndex = 0;
    for (int s = 0; s < num_species; s++) {
    if (s == inertIndex) continue;
    double own_coeff = mut_sct[index] + rhoD[num_cells * s + index];
    double sum_diag = 0;
    // lower
    for (int i = 0; i < diag_index; i++)
    {
        int neighbor_index = neighbor_offset + i;
        int neighbor_cell_id = csr_col_index[i + row_index];
        double w = weight[neighbor_index];
        double nei_coeff = mut_sct[neighbor_cell_id] + rhoD[num_cells * s + neighbor_cell_id];
        double gamma = w * (nei_coeff - own_coeff) + own_coeff;
        double gamma_magsf = gamma * magsf[neighbor_index];
        double coeff = gamma_magsf * distance[neighbor_index];
        A_csr_output[mtxIndex * (num_cells + num_faces) + row_index + i] =
            A_csr_input[mtxIndex * (num_cells + num_faces) + row_index + i] + coeff * sign;

        sum_diag += (-coeff);
    }
    // upper
    for (int i = diag_index + 1; i < row_elements; i++)
    {
        int neighbor_index = neighbor_offset + i - 1;
        int neighbor_cell_id = csr_col_index[i + row_index];
        double w = weight[neighbor_index];
        double nei_coeff = mut_sct[neighbor_cell_id] + rhoD[num_cells * s + neighbor_cell_id];
        double gamma = w * (own_coeff - nei_coeff) + nei_coeff;
        double gamma_magsf = gamma * magsf[neighbor_index];
        double coeff = gamma_magsf * distance[neighbor_index];
        A_csr_output[mtxIndex * (num_cells + num_faces) + row_index + i] =
            A_csr_input[mtxIndex * (num_cells + num_faces) + row_index + i] + coeff * sign;

        sum_diag += (-coeff);
    }
    // diag
    A_csr_output[mtxIndex * (num_cells + num_faces) + row_index + diag_index] =
        A_csr_input[mtxIndex * (num_cells + num_faces) + row_index + diag_index] + sum_diag * sign;
    ++mtxIndex;
    }
}

__global__ void fvm_laplacian_uncorrected_scalar_boundary(int num_cells, int num_faces, int num_boundary_cells, int num_boundary_faces,
        int num_species, int inertIndex,
        const int *csr_row_index, const int *csr_diag_index, const int *boundary_cell_offset,
        const int *boundary_cell_id, const double *boundary_mut_sct, const double *boundary_rhoD,
        const double *boundary_magsf, const int *bouPermedIndex,
        const double *gradient_internal_coeffs, const double *gradient_boundary_coeffs,
        const double sign, const double *A_csr_input, const double *b_input, double *A_csr_output, double *b_output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_boundary_cells)
        return;

    int cell_offset = boundary_cell_offset[index];
    int next_cell_offset = boundary_cell_offset[index + 1];
    int cell_index = boundary_cell_id[cell_offset];

    int row_index = csr_row_index[cell_index];
    int diag_index = csr_diag_index[cell_index];
    int csr_index = row_index + diag_index;

    int mtxIndex = 0;
    for (int s = 0; s < num_species; s++) {
    if (s == inertIndex) continue;
    double internal_coeffs = 0;
    double boundary_coeffs = 0;
    for (int i = cell_offset; i < next_cell_offset; i++)
    {
        int permute_index = bouPermedIndex[i];
        double gamma = boundary_mut_sct[permute_index] + boundary_rhoD[num_boundary_faces * s + permute_index];
        double gamma_magsf = gamma * boundary_magsf[i];
        internal_coeffs += gamma_magsf * gradient_internal_coeffs[num_boundary_faces * s + i];
        boundary_coeffs += gamma_magsf * gradient_boundary_coeffs[num_boundary_faces * s + i];
    }

    A_csr_output[mtxIndex * (num_cells + num_faces) + csr_index] =
        A_csr_input[mtxIndex * (num_cells + num_faces) + csr_index] + internal_coeffs * sign;
    b_output[mtxIndex * num_cells + cell_index] =
        b_input[mtxIndex * num_cells + cell_index] + boundary_coeffs * sign;
    ++mtxIndex;
    }
}

__global__ void fvc_laplacian_internal(int num_cells, int num_species,
        const int *csr_row_index, const int *csr_col_index, const int *csr_diag_index,
        const double *alpha, const double *hai, const double* y,
        const double *weight, const double *magsf, const double *distance,
        const double* volume, double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;

    // A_csr has one more element in each row: itself
    int row_index = csr_row_index[index];
    int row_elements = csr_row_index[index + 1] - row_index;
    int diag_index = csr_diag_index[index];
    int neighbor_offset = csr_row_index[index] - index;

    double vol = volume[index];
    double sum_all_species = 0;
	for (int s = 0; s < num_species; s++) {
		double own_vf = y[num_cells * s + index];
		double own_coeff = alpha[index] * hai[num_cells * s + index];
		double sum = 0;
		// lower
		for (int i = 0; i < diag_index; i++)
		{
			int neighbor_index = neighbor_offset + i;
			int neighbor_cell_id = csr_col_index[i + row_index];
			double w = weight[neighbor_index];
			double nei_vf = y[num_cells * s + neighbor_cell_id];
			double nei_coeff = alpha[neighbor_cell_id] * hai[num_cells * s + neighbor_cell_id];
			double face_gamma = (1 - w) * own_coeff + w * nei_coeff;
			double sngrad = distance[neighbor_index] * (own_vf - nei_vf);
			double value = face_gamma * sngrad * magsf[neighbor_index];
			sum -= value;
		}
		// upper
		for (int i = diag_index + 1; i < row_elements; i++)
		{
			int neighbor_index = neighbor_offset + i - 1;
			int neighbor_cell_id = csr_col_index[i + row_index];
			double w = weight[neighbor_index];
			double nei_vf = y[num_cells * s + neighbor_cell_id];
			double nei_coeff = alpha[neighbor_cell_id] * hai[num_cells * s + neighbor_cell_id];
			double face_gamma = w * own_coeff + (1 - w) * nei_coeff;
			double sngrad = distance[neighbor_index] * (nei_vf - own_vf);
			double value = face_gamma * sngrad * magsf[neighbor_index];
			sum += value;
		}
		sum_all_species += sum;
	}
	output[index] = sum_all_species / vol;
}

__global__ void yeqn_update_BoundaryCoeffs_kernel(int num_boundary_faces, int num_species,
        const double *boundary_phi, double *internal_coeffs,
                                                  double *boundary_coeffs, double *laplac_internal_coeffs,
                                                  double *laplac_boundary_coeffs, const int *U_patch_type)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_boundary_faces)
        return;

    int patchIndex = U_patch_type[index];
    for (int s = 0; s < num_species; s++) {
    double valueInternalCoeffs, valueBoundaryCoeffs, gradientInternalCoeffs, gradientBoundaryCoeffs;
    switch (patchIndex)
    {
        case 0: // zeroGradient
        {
            valueInternalCoeffs = 1.;
            valueBoundaryCoeffs = 0.;
            gradientInternalCoeffs = 0.;
            gradientBoundaryCoeffs = 0.;
            break;
        }
        // TODO implement coupled and fixedValue conditions
    }

    internal_coeffs[num_boundary_faces * s + index] = valueInternalCoeffs;
    boundary_coeffs[num_boundary_faces * s + index] = valueBoundaryCoeffs;
    laplac_internal_coeffs[num_boundary_faces * s + index] = gradientInternalCoeffs;
    laplac_boundary_coeffs[num_boundary_faces * s + index] = gradientBoundaryCoeffs;
    }
}

__global__ void yeqn_correct_BoundaryConditions_kernel(int num_cells, int num_boundary_cells, int num_boundary_faces, int num_species,
                                                       const int *boundary_cell_offset, const int *boundary_cell_id,
                                                       const double *species, double *boundary_species, const int *Y_patch_type)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_boundary_cells)
        return;

    int cell_offset = boundary_cell_offset[index];
    int next_cell_offset = boundary_cell_offset[index + 1];
    int cell_index = boundary_cell_id[cell_offset];

    for (int i = cell_offset; i < next_cell_offset; i++)
    {
        int patchIndex = Y_patch_type[i];
        switch (patchIndex)
        {
            case 0: // zeroGradient
            {
                for (int speciesID = 0; speciesID < num_species; speciesID++)
                {
                    boundary_species[speciesID * num_boundary_faces + i] = species[speciesID * num_cells + cell_index];
                }
                break;
            }
            case 1:
                break;
            // TODO implement coupled conditions
        }
    }
}

dfYEqn::dfYEqn(dfMatrixDataBase &dataBase, const std::string &modeStr, const std::string &cfgFile, const int inertIndex)
    : dataBase_(dataBase), inertIndex(inertIndex)
{
    stream = dataBase_.stream;
    num_species = dataBase_.num_species;
    num_cells = dataBase_.num_cells;
    num_faces = dataBase_.num_faces;
    num_surfaces = dataBase_.num_surfaces;
    num_boundary_cells = dataBase_.num_boundary_cells;
    num_boundary_faces = dataBase_.num_boundary_faces;
    cell_bytes = dataBase_.cell_bytes;
    boundary_face_bytes = dataBase_.boundary_face_bytes;

    YSolverSet.resize(num_species - 1); // consider inert species
    for (auto &solver : YSolverSet)
        solver = new AmgXSolver(modeStr, cfgFile);

    d_A_csr_row_index = dataBase_.d_A_csr_row_index;
    d_A_csr_diag_index = dataBase_.d_A_csr_diag_index;
    d_A_csr_col_index = dataBase_.d_A_csr_col_index;

    h_A_csr = new double[(num_cells + num_faces) * (num_species - 1)];
    h_b = new double[num_cells * (num_species - 1)];
    hipHostMalloc(&h_psi, num_cells * num_species * sizeof(double));

    checkCudaErrors(hipMalloc((void **)&d_A_csr, (num_cells + num_faces) * (num_species - 1) * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&d_b, cell_bytes * (num_species - 1)));
    checkCudaErrors(hipMalloc((void **)&d_psi, cell_bytes * (num_species - 1)));
    checkCudaErrors(hipMalloc((void **)&d_phiUc, num_faces * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&d_phiUc_boundary, num_boundary_faces * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&d_mut_Sct, cell_bytes));
    checkCudaErrors(hipMalloc((void **)&d_boundary_mut_sct, boundary_face_bytes));

    checkCudaErrors(hipMalloc((void **)&d_boundary_Y, boundary_face_bytes * num_species));

    checkCudaErrors(hipMalloc((void **)&d_hai, cell_bytes * num_species));
    checkCudaErrors(hipMalloc((void **)&d_boundary_hai, boundary_face_bytes * num_species));
    checkCudaErrors(hipMalloc((void **)&d_rhoD, cell_bytes * num_species));
    checkCudaErrors(hipMalloc((void **)&d_boundary_rhoD, boundary_face_bytes * num_species));

    checkCudaErrors(hipMalloc((void **)&d_sum_rhoD_grady, 3 * cell_bytes));
    checkCudaErrors(hipMalloc((void **)&d_sum_boundary_rhoD_grady, 3 * boundary_face_bytes));
    checkCudaErrors(hipMalloc((void **)&d_sum_hai_rhoD_grady, 3 * cell_bytes));
    checkCudaErrors(hipMalloc((void **)&d_sum_boundary_hai_rhoD_grady, 3 * boundary_face_bytes));
    checkCudaErrors(hipMalloc((void **)&d_sum_hai_y, cell_bytes));
    checkCudaErrors(hipMalloc((void **)&d_sum_boundary_hai_y, boundary_face_bytes));

    checkCudaErrors(hipMalloc((void **)&d_grady, 3 * cell_bytes * num_species));
    checkCudaErrors(hipMalloc((void **)&d_boundary_grady, 3 * boundary_face_bytes * num_species));

    checkCudaErrors(hipMalloc((void **)&d_alpha, cell_bytes));

    // zeroGradient
    checkCudaErrors(hipMemsetAsync(dataBase_.d_internal_coeffs_Y, 1, boundary_face_bytes * num_species, stream));
    checkCudaErrors(hipMemsetAsync(dataBase_.d_boundary_coeffs_Y, 0, boundary_face_bytes * num_species, stream));
    checkCudaErrors(hipMemsetAsync(dataBase_.d_laplac_internal_coeffs_Y, 0, boundary_face_bytes * num_species, stream));
    checkCudaErrors(hipMemsetAsync(dataBase_.d_laplac_boundary_coeffs_Y, 0, boundary_face_bytes * num_species, stream));
}

void dfYEqn::initializeTimeStep()
{
    // consider inert species
    // initialize matrix value
    checkCudaErrors(hipMemsetAsync(d_A_csr, 0, (num_cells + num_faces) * (num_species - 1) * sizeof(double), stream));
    checkCudaErrors(hipMemsetAsync(d_b, 0, cell_bytes * (num_species - 1), stream));
    // initialize variables in each time step
    checkCudaErrors(hipMemsetAsync(d_psi, 0, cell_bytes * (num_species - 1), stream));

    // initialize boundary coeffs
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (dataBase_.num_boundary_faces + threads_per_block - 1) / threads_per_block;
    yeqn_update_BoundaryCoeffs_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(
            num_boundary_faces, num_species,
            dataBase_.d_boundary_phi,
            dataBase_.d_internal_coeffs_Y,
            dataBase_.d_boundary_coeffs_Y,
            dataBase_.d_laplac_internal_coeffs_Y,
            dataBase_.d_laplac_boundary_coeffs_Y,
            dataBase_.d_boundary_YpatchType);
}

void dfYEqn::upwindWeight()
{
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (num_faces + threads_per_block - 1) / threads_per_block;
    getUpwindWeight<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_faces, dataBase_.d_phi, dataBase_.d_weight_upwind);
}

void dfYEqn::fvm_laplacian_and_sumYDiffError_diffAlphaD_hDiffCorrFlux(std::vector<double *> Y_old, std::vector<double *> boundary_Y,
        std::vector<const double *> hai, std::vector<double *> boundary_hai,
        std::vector<const double *> rhoD, std::vector<double *> boundary_rhoD,
        const double *mut_Sct, const double *boundary_mut_Sct, const double *alpha)
{
    // initialize variables in each time step
    checkCudaErrors(hipMemcpyAsync(d_boundary_mut_sct, boundary_mut_Sct, boundary_face_bytes, hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipMemcpyAsync(d_mut_Sct, mut_Sct, cell_bytes, hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipMemcpyAsync(d_alpha, alpha, cell_bytes, hipMemcpyHostToDevice, stream));

    checkCudaErrors(hipMemsetAsync(d_sum_rhoD_grady, 0, 3 * cell_bytes, stream));
    checkCudaErrors(hipMemsetAsync(d_sum_boundary_rhoD_grady, 0, 3 * boundary_face_bytes, stream));
    checkCudaErrors(hipMemsetAsync(d_sum_hai_rhoD_grady, 0, 3 * cell_bytes, stream));
    checkCudaErrors(hipMemsetAsync(d_sum_boundary_hai_rhoD_grady, 0, 3 * boundary_face_bytes, stream));
    checkCudaErrors(hipMemsetAsync(d_sum_hai_y, 0, cell_bytes, stream));
    checkCudaErrors(hipMemsetAsync(d_sum_boundary_hai_y, 0, boundary_face_bytes, stream));
    checkCudaErrors(hipMemsetAsync(dataBase_.d_hDiffCorrFlux, 0, 3 * cell_bytes, stream));
    checkCudaErrors(hipMemsetAsync(dataBase_.d_boundary_hDiffCorrFlux, 0, 3 * boundary_face_bytes, stream));
    checkCudaErrors(hipMemsetAsync(dataBase_.d_diffAlphaD, 0, cell_bytes, stream));

    size_t threads_per_block, blocks_per_grid;
    for (size_t i = 0; i < num_species; ++i)
    {
        checkCudaErrors(hipMemcpyAsync(dataBase_.d_Y + i * num_cells, Y_old[i], cell_bytes, hipMemcpyHostToDevice, stream));
        if (uploadBoundaryY)
        {
            checkCudaErrors(hipMemcpyAsync(d_boundary_Y + i * num_boundary_faces, boundary_Y[i], boundary_face_bytes,
                        hipMemcpyHostToDevice, stream));
        }
        checkCudaErrors(hipMemcpyAsync(d_hai + i * num_cells, hai[i], cell_bytes, hipMemcpyHostToDevice, stream));
        checkCudaErrors(hipMemcpyAsync(d_boundary_hai + i * num_boundary_faces, boundary_hai[i], boundary_face_bytes,
                    hipMemcpyHostToDevice, stream));
        checkCudaErrors(hipMemcpyAsync(d_rhoD + i * num_cells, rhoD[i], cell_bytes, hipMemcpyHostToDevice, stream));
        checkCudaErrors(hipMemcpyAsync(d_boundary_rhoD + i * num_boundary_faces, boundary_rhoD[i], boundary_face_bytes,
                    hipMemcpyHostToDevice, stream));
    }
    // fvc::grad(Yi)
    threads_per_block = 1024;
    blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    fvc_grad_internal<<<blocks_per_grid, threads_per_block, 0, stream>>>(
            num_cells, num_species,
            d_A_csr_row_index, d_A_csr_col_index, d_A_csr_diag_index,
            dataBase_.d_face_vector, dataBase_.d_weight, dataBase_.d_Y,
            dataBase_.d_volume, d_grady);
    blocks_per_grid = (num_boundary_cells + threads_per_block - 1) / threads_per_block;
    fvc_grad_boundary<<<blocks_per_grid, threads_per_block, 0, stream>>>(
            num_cells, num_boundary_cells, num_boundary_faces, num_species,
            dataBase_.d_boundary_cell_offset, dataBase_.d_boundary_cell_id, dataBase_.d_bouPermedIndex,
            dataBase_.d_boundary_face_vector, d_boundary_Y,
            dataBase_.d_volume, d_grady, d_grady, uploadBoundaryY);
    blocks_per_grid = (num_boundary_cells + threads_per_block - 1) / threads_per_block;
    correct_boundary_conditions<<<blocks_per_grid, threads_per_block, 0, stream>>>(
            num_cells, num_boundary_cells, num_boundary_faces, num_species,
            dataBase_.d_boundary_cell_offset, dataBase_.d_boundary_cell_id,
            dataBase_.d_boundary_face_vector, dataBase_.d_boundary_face,
            d_grady, d_boundary_grady);

    // sum(chemistry->hai(i)*chemistry->rhoD(i)*fvc::grad(Yi))
    // sum(chemistry->rhoD(i)*fvc::grad(Yi)), also be called sumYDiffError
    // sum(chemistry->hai(i)*Yi)
    blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    sumError_internal<<<blocks_per_grid, threads_per_block, 0, stream>>>(
            num_cells, num_species,
            d_hai, d_rhoD, dataBase_.d_Y, d_grady,
            d_sum_hai_rhoD_grady, d_sum_rhoD_grady, d_sum_hai_y);
    blocks_per_grid = (num_boundary_faces + threads_per_block - 1) / threads_per_block;
    sumError_boundary<<<blocks_per_grid, threads_per_block, 0, stream>>>(
            num_boundary_faces, num_species,
            dataBase_.d_bouPermedIndex,
            d_boundary_hai, d_boundary_rhoD, d_boundary_Y, d_boundary_grady,
            d_sum_boundary_hai_rhoD_grady, d_sum_boundary_rhoD_grady, d_sum_boundary_hai_y, uploadBoundaryY);

    // compute diffAlphaD
    // TODO non-resonable, fvc_laplacian_internal will failed if threads_per_block = 1024
    threads_per_block = 512;
    blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    fvc_laplacian_internal<<<blocks_per_grid, threads_per_block, 0, stream>>>(
            num_cells, num_species,
            d_A_csr_row_index, d_A_csr_col_index, d_A_csr_diag_index,
            d_alpha, d_hai, dataBase_.d_Y,
            dataBase_.d_weight, dataBase_.d_face, dataBase_.d_deltaCoeffs,
            dataBase_.d_volume, dataBase_.d_diffAlphaD);

    // fvm::laplacian
    // TODO non-resonable, fvm_laplacian_uncorrected_scalar_internal will failed if threads_per_block = 1024
    threads_per_block = 512;
    blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    fvm_laplacian_uncorrected_scalar_internal<<<blocks_per_grid, threads_per_block, 0, stream>>>(
            num_cells, num_faces, num_species, inertIndex,
            d_A_csr_row_index, d_A_csr_col_index, d_A_csr_diag_index,
            d_mut_Sct, d_rhoD, dataBase_.d_weight, dataBase_.d_face, dataBase_.d_deltaCoeffs,
            -1., d_A_csr, d_A_csr);
    // TODO non-resonable, fvm_laplacian_uncorrected_scalar_boundary will failed if threads_per_block = 1024
    threads_per_block = 512;
    blocks_per_grid = (num_boundary_cells + threads_per_block - 1) / threads_per_block;
    fvm_laplacian_uncorrected_scalar_boundary<<<blocks_per_grid, threads_per_block, 0, stream>>>(
            num_cells, num_faces, num_boundary_cells, num_boundary_faces,
            num_species, inertIndex,
            d_A_csr_row_index, d_A_csr_diag_index,
            dataBase_.d_boundary_cell_offset, dataBase_.d_boundary_cell_id,
            d_boundary_mut_sct, d_boundary_rhoD, dataBase_.d_boundary_face, dataBase_.d_bouPermedIndex,
            dataBase_.d_laplac_internal_coeffs_Y, dataBase_.d_laplac_boundary_coeffs_Y,
            -1., d_A_csr, d_b, d_A_csr, d_b);

    uploadBoundaryY = false;

    threads_per_block = 1024;
    blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    calculate_hDiffCorrFlux<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells,
            d_sum_hai_rhoD_grady, d_sum_rhoD_grady, d_sum_hai_y, dataBase_.d_hDiffCorrFlux);
    blocks_per_grid = (num_boundary_faces + threads_per_block - 1) / threads_per_block;
    calculate_hDiffCorrFlux<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_boundary_faces,
            d_sum_boundary_hai_rhoD_grady, d_sum_boundary_rhoD_grady, d_sum_boundary_hai_y, dataBase_.d_boundary_hDiffCorrFlux);
}

void dfYEqn::fvm_ddt()
{
    // fvm::ddt(rho, Yi)
    size_t threads_per_block, blocks_per_grid;
    threads_per_block = 1024;
    blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    fvm_ddt_kernel_scalar<<<blocks_per_grid, threads_per_block, 0, stream>>>(
            num_cells, num_faces, num_species, inertIndex,
            dataBase_.rdelta_t,
            d_A_csr_row_index, d_A_csr_diag_index,
            dataBase_.d_rho_old, dataBase_.d_rho_new, dataBase_.d_volume, dataBase_.d_Y,
            d_A_csr, d_b, d_A_csr, d_b);
}

void dfYEqn::fvm_div_phi()
{
    // mvConvection->fvmDiv(phi, Yi)
    size_t threads_per_block, blocks_per_grid;
    threads_per_block = 512;
    blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    fvm_div_internal_scalar<<<blocks_per_grid, threads_per_block, 0, stream>>>(
            num_cells, num_faces, num_species, inertIndex,
            d_A_csr_row_index, d_A_csr_diag_index, dataBase_.d_weight_upwind, dataBase_.d_phi,
            d_A_csr, d_A_csr);
    blocks_per_grid = (num_boundary_cells + threads_per_block - 1) / threads_per_block;
    fvm_div_boundary_scalar<<<blocks_per_grid, threads_per_block, 0, stream>>>(
            num_cells, num_faces, num_boundary_cells, num_boundary_faces, num_species, inertIndex,
            d_A_csr_row_index, d_A_csr_diag_index, dataBase_.d_boundary_phi,
            dataBase_.d_boundary_cell_offset, dataBase_.d_boundary_cell_id,
            dataBase_.d_internal_coeffs_Y, dataBase_.d_boundary_coeffs_Y,
            d_A_csr, d_A_csr, d_b, d_b);
}

void dfYEqn::fvm_div_phiUc()
{
    size_t threads_per_block, blocks_per_grid;

    // compue phiUc
    threads_per_block = 512;
    blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    calculate_phiUc_internal<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells,
            d_A_csr_row_index, d_A_csr_col_index, d_A_csr_diag_index,
            dataBase_.d_face_vector, dataBase_.d_weight, d_sum_rhoD_grady, d_phiUc);
    blocks_per_grid = (num_boundary_faces + threads_per_block - 1) / threads_per_block;
    calculate_phiUc_boundary<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_boundary_faces,
            dataBase_.d_boundary_cell_offset, dataBase_.d_boundary_cell_id,
            dataBase_.d_boundary_face_vector, d_sum_boundary_rhoD_grady, d_phiUc_boundary);

    // mvConvection->fvmDiv(phiUc, Yi)
    blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    fvm_div_internal_scalar<<<blocks_per_grid, threads_per_block, 0, stream>>>(
            num_cells, num_faces, num_species, inertIndex,
            d_A_csr_row_index, d_A_csr_diag_index, dataBase_.d_weight_upwind, d_phiUc,
            d_A_csr, d_A_csr);
    blocks_per_grid = (num_boundary_cells + threads_per_block - 1) / threads_per_block;
    fvm_div_boundary_scalar<<<blocks_per_grid, threads_per_block, 0, stream>>>(
            num_cells, num_faces, num_boundary_cells, num_boundary_faces, num_species, inertIndex,
            d_A_csr_row_index, d_A_csr_diag_index, d_phiUc_boundary,
            dataBase_.d_boundary_cell_offset, dataBase_.d_boundary_cell_id,
            dataBase_.d_internal_coeffs_Y, dataBase_.d_boundary_coeffs_Y,
            d_A_csr, d_A_csr, d_b, d_b);
}

void dfYEqn::checkValue(bool print, char *filename)
{
    checkCudaErrors(hipMemcpyAsync(h_A_csr, d_A_csr, (num_cells + num_faces) * sizeof(double), hipMemcpyDeviceToHost, stream));
    checkCudaErrors(hipMemcpyAsync(h_b, d_b, num_cells * sizeof(double), hipMemcpyDeviceToHost, stream));

    // Synchronize stream
    checkCudaErrors(hipStreamSynchronize(stream));
    if (print)
    {
        for (int i = 0; i < (num_faces + num_cells); i++)
            fprintf(stderr, "h_A_csr[%d]: %.15lf\n", i, h_A_csr[i]);
        for (int i = 0; i < num_cells; i++)
            fprintf(stderr, "h_b[%d]: %.15lf\n", i, h_b[i]);
    }

    char *input_file = filename;
    FILE *fp = fopen(input_file, "rb+");
    if (fp == NULL)
    {
        fprintf(stderr, "Failed to open input file: %s!\n", input_file);
    }

    int readfile = 0;
    double *of_b = new double[num_cells];
    double *of_A = new double[num_faces + num_cells];
    readfile = fread(of_b, num_cells * sizeof(double), 1, fp);
    readfile = fread(of_A, (num_faces + num_cells) * sizeof(double), 1, fp);

    std::vector<double> h_A_of_vec_1mtx(num_faces + num_cells, 0);
    for (int i = 0; i < num_faces + num_cells; i++)
    {
        h_A_of_vec_1mtx[i] = of_A[dataBase_.tmpPermutatedList[i]];
    }
    if (print)
    {
        for (int i = 0; i < (num_faces + num_cells); i++)
            printf("h_A_of_vec_1mtx[%d]: %.15lf\n", i, h_A_of_vec_1mtx[i]);
        for (int i = 0; i < num_cells; i++)
            printf("h_b_of_vec[%d]: %.15lf\n", i, of_b[i]);
    }

    fprintf(stderr, "check of h_A_csr\n");
    checkVectorEqual(num_faces + num_cells, h_A_of_vec_1mtx.data(), h_A_csr, 1e-5);
    fprintf(stderr, "check of h_b\n");
    checkVectorEqual(num_cells, of_b, h_b, 1e-5);
}

void dfYEqn::solve()
{
    checkCudaErrors(hipStreamSynchronize(stream));

    int nNz = num_cells + num_faces; // matrix entries
    if (num_iteration == 0)          // first interation
    {
        printf("Initializing AmgX Linear Solver\n");
        int solverIndex = 0;
        for (auto &solver : YSolverSet)
        {
            solver->setOperator(num_cells, nNz, d_A_csr_row_index, d_A_csr_col_index, d_A_csr + solverIndex * nNz);
            ++solverIndex;
        }
    }
    else
    {
        int solverIndex = 0;
        for (auto &solver : YSolverSet)
        {
            solver->updateOperator(num_cells, nNz, d_A_csr + solverIndex * nNz);
            ++solverIndex;
        }
    }
    int mtxIndex = 0;
    for (size_t i = 0; i < num_species; ++i)
    {
        if (i == inertIndex)
            continue;

        YSolverSet[mtxIndex]->solve(num_cells, dataBase_.d_Y + i * num_cells, d_b + mtxIndex * num_cells);
        ++mtxIndex;
    }

    size_t threads_per_block, blocks_per_grid;
    threads_per_block = 1024;
    blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    compute_inertIndex_y<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, num_species, inertIndex, dataBase_.d_Y);
    checkCudaErrors(hipMemcpyAsync(h_psi, dataBase_.d_Y, num_species * cell_bytes, hipMemcpyDeviceToHost, stream));

    num_iteration++;
    // checkCudaErrors(hipStreamSynchronize(stream));
    // for (size_t i = 0; i < num_cells; i++)
    //     fprintf(stderr, "h_species_gpu[%d]: %.5e\n", i, h_psi[i + 0 * num_cells]);
}

void dfYEqn::sync()
{
    checkCudaErrors(hipStreamSynchronize(stream));
}

void dfYEqn::updatePsi(double *Psi, int speciesIndex)
{
    checkCudaErrors(hipStreamSynchronize(stream));
    memcpy(Psi, h_psi + speciesIndex * num_cells, cell_bytes);
}

void dfYEqn::correctBoundaryConditions()
{
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (num_boundary_cells + threads_per_block - 1) / threads_per_block;
    yeqn_correct_BoundaryConditions_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, num_boundary_cells, num_boundary_faces, num_species,
                                                                                              dataBase_.d_boundary_cell_offset, dataBase_.d_boundary_cell_id,
                                                                                              dataBase_.d_Y, d_boundary_Y, dataBase_.d_boundary_YpatchType);
    // double *h_boundary_Y = new double[num_boundary_faces];
    // hipMemcpy(h_boundary_Y, d_boundary_Y, num_boundary_faces * sizeof(double), hipMemcpyDeviceToHost);
    // for (int i = 0; i < num_boundary_faces; i++)
    // {
    //     printf("h_boundary_GPU[%d] = %e\n", i, h_boundary_Y[i]);
    // }
}

dfYEqn::~dfYEqn()
{
}
