#include "hip/hip_runtime.h"
#include "dfMatrixOpBase.H"
#include "dfMatrixDataBase.H"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void permute_vector_d2h_kernel(int num_cells, const double *input, double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;

    output[index * 3 + 0] = input[num_cells * 0 + index];
    output[index * 3 + 1] = input[num_cells * 1 + index];
    output[index * 3 + 2] = input[num_cells * 2 + index];
}

__global__ void permute_vector_h2d_kernel(int num_cells, const double *input, double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;

    output[num_cells * 0 + index] = input[index * 3 + 0];
    output[num_cells * 1 + index] = input[index * 3 + 1];
    output[num_cells * 2 + index] = input[index * 3 + 2];
}

__global__ void field_multiply_scalar_kernel(int num_cells, int num_boundary_surfaces,
        const double *input1, const double *input2, double *output,
        const double *boundary_input1, const double *boundary_input2, double *boundary_output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < num_cells) {
        output[index] = input1[index] * input2[index];
    }
    if (index < num_boundary_surfaces) {
        boundary_output[index] = boundary_input1[index] * boundary_input2[index];
    }
}

__global__ void fvc_to_source_vector_kernel(int num_cells, const double *volume, const double *fvc_output, double *source)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;

    source[index * 3 + 0] += fvc_output[index * 3 + 0] * volume[index];
    source[index * 3 + 1] += fvc_output[index * 3 + 1] * volume[index];
    source[index * 3 + 2] += fvc_output[index * 3 + 2] * volume[index];
}

__global__ void update_boundary_coeffs_zeroGradient_vector(int num, int offset,
        double *value_internal_coeffs, double *value_boundary_coeffs,
        double *gradient_internal_coeffs, double *gradient_boundary_coeffs)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;

    int start_index = offset + index;
    // valueInternalCoeffs = 1
    // valueBoundaryCoeffs = 0
    // gradientInternalCoeffs = 0
    // gradientBoundaryCoeffs = 0
    value_internal_coeffs[start_index * 3 + 0] = 1;
    value_internal_coeffs[start_index * 3 + 1] = 1;
    value_internal_coeffs[start_index * 3 + 2] = 1;
    value_boundary_coeffs[start_index * 3 + 0] = 0;
    value_boundary_coeffs[start_index * 3 + 1] = 0;
    value_boundary_coeffs[start_index * 3 + 2] = 0;
    gradient_internal_coeffs[start_index * 3 + 0] = 0;
    gradient_internal_coeffs[start_index * 3 + 1] = 0;
    gradient_internal_coeffs[start_index * 3 + 2] = 0;
    gradient_boundary_coeffs[start_index * 3 + 0] = 0;
    gradient_boundary_coeffs[start_index * 3 + 1] = 0;
    gradient_boundary_coeffs[start_index * 3 + 2] = 0;
}

__global__ void scale_dev2t_tensor_kernel(int num, const double *vf1, double *vf2)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;

    double scale = vf1[index];
    double val_xx = vf2[index * 9 + 0];
    double val_xy = vf2[index * 9 + 1];
    double val_xz = vf2[index * 9 + 2];
    double val_yx = vf2[index * 9 + 3];
    double val_yy = vf2[index * 9 + 4];
    double val_yz = vf2[index * 9 + 5];
    double val_zx = vf2[index * 9 + 6];
    double val_zy = vf2[index * 9 + 7];
    double val_zz = vf2[index * 9 + 8];
    double trace_coeff = (2. / 3.) * (val_xx + val_yy + val_zz);
    vf2[index * 9 + 0] = scale * (val_xx - trace_coeff);
    vf2[index * 9 + 1] = scale * val_yx;
    vf2[index * 9 + 2] = scale * val_zx;
    vf2[index * 9 + 3] = scale * val_xy;
    vf2[index * 9 + 4] = scale * (val_yy - trace_coeff);
    vf2[index * 9 + 5] = scale * val_zy;
    vf2[index * 9 + 6] = scale * val_xz;
    vf2[index * 9 + 7] = scale * val_yz;
    vf2[index * 9 + 8] = scale * (val_zz - trace_coeff);
}

__global__ void fvm_ddt_vector_kernel(int num_cells, double rDeltaT,
        const double *rho, const double *rho_old, const double *vf, const double *volume,
        double *diag, double *source, double sign)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;

    diag[index] += rDeltaT * rho[index] * volume[index] * sign;
    // TODO: skip moving
    source[index * 3 + 0] += rDeltaT * rho_old[index] * vf[index * 3 + 0] * volume[index] * sign;
    source[index * 3 + 1] += rDeltaT * rho_old[index] * vf[index * 3 + 1] * volume[index] * sign;
    source[index * 3 + 2] += rDeltaT * rho_old[index] * vf[index * 3 + 2] * volume[index] * sign;
}

__global__ void fvm_div_vector_internal(int num_surfaces,
        const int *lower_index, const int *upper_index,
        const double *phi, const double *weight,
        double *lower, double *upper, double *diag, double sign)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_surfaces)
        return;

    double w = weight[index];
    double f = phi[index];

    double lower_value = (-w) * f * sign;
    double upper_value = (1 - w) * f * sign;
    lower[index] += lower_value;
    upper[index] += upper_value;
    // if (index == 0) printf("index = 0, lower: %.16lf, upper:%.16lf\n", lower[index], upper[index]);

    int owner = lower_index[index];
    int neighbor = upper_index[index];
    atomicAdd(&(diag[owner]), -lower_value);
    atomicAdd(&(diag[neighbor]), -upper_value);
}

__global__ void fvm_div_vector_boundary(int num, int offset,
        const double *boundary_phi, const double *value_internal_coeffs, const double *value_boundary_coeffs,
        double *internal_coeffs, double *boundary_coeffs, double sign)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;

    int start_index = offset + index;
    double boundary_f = boundary_phi[start_index];
    internal_coeffs[start_index * 3 + 0] += boundary_f * value_internal_coeffs[start_index * 3 + 0] * sign;
    internal_coeffs[start_index * 3 + 1] += boundary_f * value_internal_coeffs[start_index * 3 + 1] * sign;
    internal_coeffs[start_index * 3 + 2] += boundary_f * value_internal_coeffs[start_index * 3 + 2] * sign;
    boundary_coeffs[start_index * 3 + 0] += boundary_f * value_boundary_coeffs[start_index * 3 + 0] * sign;
    boundary_coeffs[start_index * 3 + 1] += boundary_f * value_boundary_coeffs[start_index * 3 + 1] * sign;
    boundary_coeffs[start_index * 3 + 2] += boundary_f * value_boundary_coeffs[start_index * 3 + 2] * sign;
}

__global__ void fvm_laplacian_vector_internal(int num_surfaces,
        const int *lower_index, const int *upper_index,
        const double *weight, const double *mag_sf, const double *delta_coeffs, const double *gamma,
        double *lower, double *upper, double *diag, double sign)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_surfaces)
        return;

    int owner = lower_index[index];
    int neighbor = upper_index[index];

    double w = weight[index];
    double upper_face_gamma = w * gamma[owner] + (1 - w) * gamma[neighbor];
    double upper_value = upper_face_gamma * mag_sf[index] * delta_coeffs[index];

    // laplacian doesn't use the original lower, but use lower = upper
    //double lower_face_gamma = w * gamma[neighbor] + (1 - w) * gamma[owner];
    //double lower_value = lower_face_gamma * mag_sf[index] * delta_coeffs[index];
    double lower_value = upper_value;

    lower_value = lower_value * sign;
    upper_value = upper_value * sign;

    lower[index] += lower_value;
    upper[index] += upper_value;

    atomicAdd(&(diag[owner]), -lower_value);
    atomicAdd(&(diag[neighbor]), -upper_value);
}

__global__ void fvm_laplacian_vector_boundary(int num, int offset,
        const double *boundary_mag_sf, const double *boundary_gamma,
        const double *gradient_internal_coeffs, const double *gradient_boundary_coeffs,
        double *internal_coeffs, double *boundary_coeffs, double sign)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;

    int start_index = offset + index;
    double boundary_value = boundary_gamma[start_index] * boundary_mag_sf[start_index];
    internal_coeffs[start_index * 3 + 0] += boundary_value * gradient_internal_coeffs[start_index * 3 + 0] * sign;
    internal_coeffs[start_index * 3 + 1] += boundary_value * gradient_internal_coeffs[start_index * 3 + 1] * sign;
    internal_coeffs[start_index * 3 + 2] += boundary_value * gradient_internal_coeffs[start_index * 3 + 2] * sign;
    boundary_coeffs[start_index * 3 + 0] += boundary_value * gradient_boundary_coeffs[start_index * 3 + 0] * sign;
    boundary_coeffs[start_index * 3 + 1] += boundary_value * gradient_boundary_coeffs[start_index * 3 + 1] * sign;
    boundary_coeffs[start_index * 3 + 2] += boundary_value * gradient_boundary_coeffs[start_index * 3 + 2] * sign;
}

__global__ void fvc_ddt_scalar_kernel(int num_cells, double rDeltaT,
        const double *rho, const double *rho_old, const double *vf, const double *vf_old,
        double *output, double sign)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;
    /*
    // workaround way1 (use printf):
    double val_new = rho[index] * vf[index];
    double val_old = rho_old[index] * vf_old[index];
    // TODO: skip moving
    // TODO: wyr
    // for the case of rho = rho_old and vf = vf_old, the floating-point numerical problem will be exposed.
    // it expect zero as output, but the gpu result get a sub-normal minimal value for (val_new - val_old),
    // which smaller than 1e-16, and then enlarged by rDeltaT (1e6)
    // then the comparison of cpu result and gpu result will failed with relative error: inf,
    // e.g.:
    // cpu data: 0.0000000000000000, gpu data: 0.0000000000298050, relative error: inf
    // if I add the print line for intermediate variables of val_new and val_old, the problem disappears.
    // It seems that print line will change the compiler behavior, maybe avoiding the fma optimization of compiler.
    if (index == -1) printf("index = 0, val_new: %.40lf, val_old: %.40lf\n", val_new, val_old);
    output[index] += rDeltaT * (val_new - val_old);
    */
    /*
    // workaround way2 (use volatile):
    // volatile will change the compiler behavior, maybe avoiding the fma optimization of compiler.
    volatile double val_new = rho[index] * vf[index];
    volatile double val_old = rho_old[index] * vf_old[index];
    output[index] += rDeltaT * (val_new - val_old);
    */
    // workaround way3 (use nvcc option -fmad=false)
    output[index] += rDeltaT * (rho[index] * vf[index] - rho_old[index] * vf_old[index]) * sign;
}

__global__ void fvc_grad_vector_internal(int num_surfaces, 
        const int *lower_index, const int *upper_index, const double *face_vector,
        const double *weight, const double *field_vector, 
        double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_surfaces)
        return;
    
    double w = weight[index];
    double Sfx = face_vector[index * 3 + 0];
    double Sfy = face_vector[index * 3 + 1];
    double Sfz = face_vector[index * 3 + 2];

    int owner = lower_index[index];
    int neighbor = upper_index[index];

    double ssfx = (w * (field_vector[owner * 3 + 0] - field_vector[neighbor * 3 + 0]) + field_vector[neighbor * 3 + 0]);
    double ssfy = (w * (field_vector[owner * 3 + 1] - field_vector[neighbor * 3 + 1]) + field_vector[neighbor * 3 + 1]);
    double ssfz = (w * (field_vector[owner * 3 + 2] - field_vector[neighbor * 3 + 2]) + field_vector[neighbor * 3 + 2]);    

    double grad_xx = Sfx * ssfx;
    double grad_xy = Sfx * ssfy;
    double grad_xz = Sfx * ssfz;
    double grad_yx = Sfy * ssfx;
    double grad_yy = Sfy * ssfy;
    double grad_yz = Sfy * ssfz;
    double grad_zx = Sfz * ssfx;
    double grad_zy = Sfz * ssfy;
    double grad_zz = Sfz * ssfz;

    // owner
    atomicAdd(&(output[owner * 9 + 0]), grad_xx);
    atomicAdd(&(output[owner * 9 + 1]), grad_xy);
    atomicAdd(&(output[owner * 9 + 2]), grad_xz);
    atomicAdd(&(output[owner * 9 + 3]), grad_yx);
    atomicAdd(&(output[owner * 9 + 4]), grad_yy);
    atomicAdd(&(output[owner * 9 + 5]), grad_yz);
    atomicAdd(&(output[owner * 9 + 6]), grad_zx);
    atomicAdd(&(output[owner * 9 + 7]), grad_zy);
    atomicAdd(&(output[owner * 9 + 8]), grad_zz);

    // neighbour
    atomicAdd(&(output[neighbor * 9 + 0]), -grad_xx);
    atomicAdd(&(output[neighbor * 9 + 1]), -grad_xy);
    atomicAdd(&(output[neighbor * 9 + 2]), -grad_xz);
    atomicAdd(&(output[neighbor * 9 + 3]), -grad_yx);
    atomicAdd(&(output[neighbor * 9 + 4]), -grad_yy);
    atomicAdd(&(output[neighbor * 9 + 5]), -grad_yz);
    atomicAdd(&(output[neighbor * 9 + 6]), -grad_zx);
    atomicAdd(&(output[neighbor * 9 + 7]), -grad_zy);
    atomicAdd(&(output[neighbor * 9 + 8]), -grad_zz);
}

// update boundary of interpolation field
// calculate the grad field
// TODO: this function is implemented for uncoupled boundary conditions
//       so it should use the more specific func name
__global__ void fvc_grad_vector_boundary(int num, int offset, const int *face2Cells,
        const double *boundary_face_vector, const double *boundary_field_vector, double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;

    int start_index = offset + index;

    double bouSfx = boundary_face_vector[start_index * 3 + 0];
    double bouSfy = boundary_face_vector[start_index * 3 + 1];
    double bouSfz = boundary_face_vector[start_index * 3 + 2];

    double boussfx = boundary_field_vector[start_index * 3 + 0];
    double boussfy = boundary_field_vector[start_index * 3 + 1];
    double boussfz = boundary_field_vector[start_index * 3 + 2];

    int cellIndex = face2Cells[start_index];

    double grad_xx = bouSfx * boussfx;
    double grad_xy = bouSfx * boussfy;
    double grad_xz = bouSfx * boussfz;
    double grad_yx = bouSfy * boussfx;
    double grad_yy = bouSfy * boussfy;
    double grad_yz = bouSfy * boussfz;
    double grad_zx = bouSfz * boussfx;
    double grad_zy = bouSfz * boussfy;
    double grad_zz = bouSfz * boussfz;

    atomicAdd(&(output[cellIndex * 9 + 0]), grad_xx);
    atomicAdd(&(output[cellIndex * 9 + 1]), grad_xy);
    atomicAdd(&(output[cellIndex * 9 + 2]), grad_xz);
    atomicAdd(&(output[cellIndex * 9 + 3]), grad_yx);
    atomicAdd(&(output[cellIndex * 9 + 4]), grad_yy);
    atomicAdd(&(output[cellIndex * 9 + 5]), grad_yz);
    atomicAdd(&(output[cellIndex * 9 + 6]), grad_zx);
    atomicAdd(&(output[cellIndex * 9 + 7]), grad_zy);
    atomicAdd(&(output[cellIndex * 9 + 8]), grad_zz);
}

__global__ void fvc_grad_scalar_internal(int num_cells, int num_surfaces,
        const int *lower_index, const int *upper_index, const double *face_vector, 
        const double *weight, const double *vf, double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_surfaces)
        return;
    
    double w = weight[index];
    double Sfx = face_vector[index * 3 + 0];
    double Sfy = face_vector[index * 3 + 1];
    double Sfz = face_vector[index * 3 + 2];

    int owner = lower_index[index];
    int neighbor = upper_index[index];

    double ssf = (w * (vf[owner] - vf[neighbor]) + vf[neighbor]);

    double grad_x = Sfx * ssf;
    double grad_y = Sfy * ssf;
    double grad_z = Sfz * ssf;

    // // owner
    // atomicAdd(&(output[num_cells * 0 + owner]), grad_x);
    // atomicAdd(&(output[num_cells * 1 + owner]), grad_y);
    // atomicAdd(&(output[num_cells * 2 + owner]), grad_z);

    // // neighbour
    // atomicAdd(&(output[num_cells * 0 + neighbor]), -grad_x);
    // atomicAdd(&(output[num_cells * 1 + neighbor]), -grad_y);
    // atomicAdd(&(output[num_cells * 2 + neighbor]), -grad_z);

    // owner
    atomicAdd(&(output[owner * 3 + 0]), grad_x);
    atomicAdd(&(output[owner * 3 + 1]), grad_y);
    atomicAdd(&(output[owner * 3 + 2]), grad_z);

    // neighbour
    atomicAdd(&(output[neighbor * 3 + 0]), -grad_x);
    atomicAdd(&(output[neighbor * 3 + 1]), -grad_y);
    atomicAdd(&(output[neighbor * 3 + 2]), -grad_z);
    
}

__global__ void fvc_grad_scalar_boundary(int num_cells, int num, int offset, const int *face2Cells,
        const double *boundary_face_vector, const double *boundary_vf, double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;

    int start_index = offset + index;

    double bouvf = boundary_vf[start_index];
    double bouSfx = boundary_face_vector[start_index * 3 + 0];
    double bouSfy = boundary_face_vector[start_index * 3 + 1];
    double bouSfz = boundary_face_vector[start_index * 3 + 2];

    int cellIndex = face2Cells[start_index];

    double grad_x = bouSfx * bouvf;
    double grad_y = bouSfy * bouvf;
    double grad_z = bouSfz * bouvf;

    atomicAdd(&(output[cellIndex * 3 + 0]), grad_x);
    atomicAdd(&(output[cellIndex * 3 + 1]), grad_y);
    atomicAdd(&(output[cellIndex * 3 + 2]), grad_z);

    // if (cellIndex == 5)
    // {
    //     printf("Sfx = %.10e, ssf = %.10e\n", bouSfx, bouvf);
    //     printf("gradx = %.10e, output = %.10e\n\n", grad_x, output[5]);
    // }
}

__global__ void divide_cell_volume_tsr(int num_cells, const double* volume, double *output, double sign)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;
    
    double vol = volume[index];
    output[index * 9 + 0] = output[index * 9 + 0] / vol * sign;
    output[index * 9 + 1] = output[index * 9 + 1] / vol * sign;
    output[index * 9 + 2] = output[index * 9 + 2] / vol * sign;
    output[index * 9 + 3] = output[index * 9 + 3] / vol * sign;
    output[index * 9 + 4] = output[index * 9 + 4] / vol * sign;
    output[index * 9 + 5] = output[index * 9 + 5] / vol * sign;
    output[index * 9 + 6] = output[index * 9 + 6] / vol * sign;
    output[index * 9 + 7] = output[index * 9 + 7] / vol * sign;
    output[index * 9 + 8] = output[index * 9 + 8] / vol * sign;
}

__global__ void divide_cell_volume_vec(int num_cells, const double* volume, double *output, double sign)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;
    
    double vol = volume[index];

    output[index * 3 + 0] = output[index * 3 + 0] / vol * sign;
    output[index * 3 + 1] = output[index * 3 + 1] / vol * sign;
    output[index * 3 + 2] = output[index * 3 + 2] / vol * sign;
}

__global__ void divide_cell_volume_scalar(int num_cells, const double* volume, double *output, double sign)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;
    
    double vol = volume[index];

    output[index] = output[index] / vol * sign;
}

__global__ void fvc_grad_vector_correctBC_zeroGradient(int num, int offset, const int *face2Cells, 
        const double *internal_grad, const double *vf, const double *boundary_sf,
        const double *boundary_mag_sf, double *boundary_grad, double sign)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;

    int start_index = offset + index;

    int cellIndex = face2Cells[start_index];

    double grad_xx = internal_grad[cellIndex * 9 + 0];
    double grad_xy = internal_grad[cellIndex * 9 + 1];
    double grad_xz = internal_grad[cellIndex * 9 + 2];
    double grad_yx = internal_grad[cellIndex * 9 + 3];
    double grad_yy = internal_grad[cellIndex * 9 + 4];
    double grad_yz = internal_grad[cellIndex * 9 + 5];
    double grad_zx = internal_grad[cellIndex * 9 + 6];
    double grad_zy = internal_grad[cellIndex * 9 + 7];
    double grad_zz = internal_grad[cellIndex * 9 + 8];

    double vfx = vf[cellIndex * 3 + 0];
    double vfy = vf[cellIndex * 3 + 1];
    double vfz = vf[cellIndex * 3 + 2];

    double n_x = boundary_sf[cellIndex * 3 + 0] / boundary_mag_sf[cellIndex];
    double n_y = boundary_sf[cellIndex * 3 + 1] / boundary_mag_sf[cellIndex];
    double n_z = boundary_sf[cellIndex * 3 + 2] / boundary_mag_sf[cellIndex];
    
    double grad_correction_x = - (n_x * grad_xx + n_y * grad_yx + n_z * grad_zx); // sn_grad_x = 0
    double grad_correction_y = - (n_x * grad_xy + n_y * grad_yy + n_z * grad_zy);
    double grad_correction_z = - (n_x * grad_xz + n_y * grad_yz + n_z * grad_zz);

    boundary_grad[start_index * 9 + 0] = (grad_xx + n_x * grad_correction_x) * sign;
    boundary_grad[start_index * 9 + 1] = (grad_xy + n_x * grad_correction_y) * sign;
    boundary_grad[start_index * 9 + 2] = (grad_xz + n_x * grad_correction_z) * sign;
    boundary_grad[start_index * 9 + 3] = (grad_yx + n_y * grad_correction_x) * sign;
    boundary_grad[start_index * 9 + 4] = (grad_yy + n_y * grad_correction_y) * sign;
    boundary_grad[start_index * 9 + 5] = (grad_yz + n_y * grad_correction_z) * sign;
    boundary_grad[start_index * 9 + 6] = (grad_zx + n_z * grad_correction_x) * sign;
    boundary_grad[start_index * 9 + 7] = (grad_zy + n_z * grad_correction_y) * sign;
    boundary_grad[start_index * 9 + 8] = (grad_zz + n_z * grad_correction_z) * sign;
}

__global__ void fvc_grad_vector_correctBC_fixedValue(int num, int offset, const int *face2Cells, 
        const double *internal_grad, const double *vf, const double *boundary_sf,
        const double *boundary_mag_sf, double *boundary_grad,
        const double *boundary_deltaCoeffs, const double *boundary_vf, double sign)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;

    int start_index = offset + index;

    int cellIndex = face2Cells[start_index];

    double grad_xx = internal_grad[cellIndex * 9 + 0];
    double grad_xy = internal_grad[cellIndex * 9 + 1];
    double grad_xz = internal_grad[cellIndex * 9 + 2];
    double grad_yx = internal_grad[cellIndex * 9 + 3];
    double grad_yy = internal_grad[cellIndex * 9 + 4];
    double grad_yz = internal_grad[cellIndex * 9 + 5];
    double grad_zx = internal_grad[cellIndex * 9 + 6];
    double grad_zy = internal_grad[cellIndex * 9 + 7];
    double grad_zz = internal_grad[cellIndex * 9 + 8];

    double vfx = vf[cellIndex * 3 + 0];
    double vfy = vf[cellIndex * 3 + 1];
    double vfz = vf[cellIndex * 3 + 2];

    double n_x = boundary_sf[start_index * 3 + 0] / boundary_mag_sf[start_index];
    double n_y = boundary_sf[start_index * 3 + 1] / boundary_mag_sf[start_index];
    double n_z = boundary_sf[start_index * 3 + 2] / boundary_mag_sf[start_index];
    
    // sn_grad: solving according to fixedValue BC
    double sn_grad_x = boundary_deltaCoeffs[start_index] * (boundary_vf[start_index * 3 + 0] - vf[cellIndex * 3 + 0]);
    double sn_grad_y = boundary_deltaCoeffs[start_index] * (boundary_vf[start_index * 3 + 1] - vf[cellIndex * 3 + 1]);
    double sn_grad_z = boundary_deltaCoeffs[start_index] * (boundary_vf[start_index * 3 + 2] - vf[cellIndex * 3 + 2]);

    double grad_correction_x = sn_grad_x - (n_x * grad_xx + n_y * grad_yx + n_z * grad_zx); // sn_grad_x = 0
    double grad_correction_y = sn_grad_y - (n_x * grad_xy + n_y * grad_yy + n_z * grad_zy);
    double grad_correction_z = sn_grad_z - (n_x * grad_xz + n_y * grad_yz + n_z * grad_zz);

    boundary_grad[start_index * 9 + 0] = (grad_xx + n_x * grad_correction_x) * sign;
    boundary_grad[start_index * 9 + 1] = (grad_xy + n_x * grad_correction_y) * sign;
    boundary_grad[start_index * 9 + 2] = (grad_xz + n_x * grad_correction_z) * sign;
    boundary_grad[start_index * 9 + 3] = (grad_yx + n_y * grad_correction_x) * sign;
    boundary_grad[start_index * 9 + 4] = (grad_yy + n_y * grad_correction_y) * sign;
    boundary_grad[start_index * 9 + 5] = (grad_yz + n_y * grad_correction_z) * sign;
    boundary_grad[start_index * 9 + 6] = (grad_zx + n_z * grad_correction_x) * sign;
    boundary_grad[start_index * 9 + 7] = (grad_zy + n_z * grad_correction_y) * sign;
    boundary_grad[start_index * 9 + 8] = (grad_zz + n_z * grad_correction_z) * sign;
}

__global__ void fvc_div_surface_scalar_internal(int num_surfaces, 
        const int *lower_index, const int *upper_index, const double *ssf,
        double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_surfaces)
        return;

    int owner = lower_index[index];
    int neighbor = upper_index[index];

    double issf = ssf[index];

    // owner
    atomicAdd(&(output[owner]), issf);

    // neighbor
    atomicAdd(&(output[neighbor]), -issf);
}

__global__ void fvc_div_surface_scalar_boundary(int num_boundary_face, const int *face2Cells,
        const double *boundary_ssf, double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_boundary_face)
        return;
    
    int cellIndex = face2Cells[index];

    atomicAdd(&(output[cellIndex]), boundary_ssf[index]);
}

__global__ void fvc_div_cell_vector_internal(int num_surfaces, 
        const int *lower_index, const int *upper_index,
        const double *field_vector, const double *weight, const double *face_vector,
        double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_surfaces)
        return;

    double w = weight[index];
    double Sfx = face_vector[index * 3 + 0];
    double Sfy = face_vector[index * 3 + 1];
    double Sfz = face_vector[index * 3 + 2];

    int owner = lower_index[index];
    int neighbor = upper_index[index];

    double ssfx = (w * (field_vector[owner * 3 + 0] - field_vector[neighbor * 3 + 0]) + field_vector[neighbor * 3 + 0]);
    double ssfy = (w * (field_vector[owner * 3 + 1] - field_vector[neighbor * 3 + 1]) + field_vector[neighbor * 3 + 1]);
    double ssfz = (w * (field_vector[owner * 3 + 2] - field_vector[neighbor * 3 + 2]) + field_vector[neighbor * 3 + 2]);

    double div = Sfx * ssfx + Sfy * ssfy + Sfz * ssfz;

    // owner
    atomicAdd(&(output[owner]), div);

    // neighbour
    atomicAdd(&(output[neighbor]), -div);
}

__global__ void fvc_div_cell_vector_boundary(int num, int offset, const int *face2Cells,
        const double *boundary_face_vector, const double *boundary_field_vector, double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;

    int start_index = offset + index;

    double bouSfx = boundary_face_vector[start_index * 3 + 0];
    double bouSfy = boundary_face_vector[start_index * 3 + 1];
    double bouSfz = boundary_face_vector[start_index * 3 + 2];

    double boussfx = boundary_field_vector[start_index * 3 + 0];
    double boussfy = boundary_field_vector[start_index * 3 + 1];
    double boussfz = boundary_field_vector[start_index * 3 + 2];

    int cellIndex = face2Cells[start_index];

    double bouDiv = bouSfx * boussfx + bouSfy * boussfy + bouSfz * boussfz;

    atomicAdd(&(output[cellIndex]), bouDiv);

}

__global__ void fvc_div_cell_tensor_internal(int num_surfaces,
        const int *lower_index, const int *upper_index,
        const double *vf, const double *weight, const double *face_vector,
        double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_surfaces)
        return;

    double w = weight[index];
    double Sfx = face_vector[index * 3 + 0];
    double Sfy = face_vector[index * 3 + 1];
    double Sfz = face_vector[index * 3 + 2];
    int owner = lower_index[index];
    int neighbor = upper_index[index];

    double ssf_xx = (w * (vf[owner * 9 + 0] - vf[neighbor * 9 + 0]) + vf[neighbor * 9 + 0]);
    double ssf_xy = (w * (vf[owner * 9 + 1] - vf[neighbor * 9 + 1]) + vf[neighbor * 9 + 1]);
    double ssf_xz = (w * (vf[owner * 9 + 2] - vf[neighbor * 9 + 2]) + vf[neighbor * 9 + 2]);
    double ssf_yx = (w * (vf[owner * 9 + 3] - vf[neighbor * 9 + 3]) + vf[neighbor * 9 + 3]);
    double ssf_yy = (w * (vf[owner * 9 + 4] - vf[neighbor * 9 + 4]) + vf[neighbor * 9 + 4]);
    double ssf_yz = (w * (vf[owner * 9 + 5] - vf[neighbor * 9 + 5]) + vf[neighbor * 9 + 5]);
    double ssf_zx = (w * (vf[owner * 9 + 6] - vf[neighbor * 9 + 6]) + vf[neighbor * 9 + 6]);
    double ssf_zy = (w * (vf[owner * 9 + 7] - vf[neighbor * 9 + 7]) + vf[neighbor * 9 + 7]);
    double ssf_zz = (w * (vf[owner * 9 + 8] - vf[neighbor * 9 + 8]) + vf[neighbor * 9 + 8]);
    double div_x = Sfx * ssf_xx + Sfy * ssf_xy + Sfz * ssf_xz;
    double div_y = Sfx * ssf_yx + Sfy * ssf_yy + Sfz * ssf_yz;
    double div_z = Sfx * ssf_zx + Sfy * ssf_zy + Sfz * ssf_zz;

    // owner
    atomicAdd(&(output[owner * 3 + 0]), div_x);
    atomicAdd(&(output[owner * 3 + 1]), div_y);
    atomicAdd(&(output[owner * 3 + 2]), div_z);

    // neighbour
    atomicAdd(&(output[neighbor * 3 + 0]), -div_x);
    atomicAdd(&(output[neighbor * 3 + 1]), -div_y);
    atomicAdd(&(output[neighbor * 3 + 2]), -div_z);
}

__global__ void fvc_div_cell_tensor_boundary(int num, int offset, const int *face2Cells,
        const double *boundary_face_vector, const double *boundary_vf, double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;

    int start_index = offset + index;

    double bouSfx = boundary_face_vector[start_index * 3 + 0];
    double bouSfy = boundary_face_vector[start_index * 3 + 1];
    double bouSfz = boundary_face_vector[start_index * 3 + 2];

    double boussf_xx = boundary_vf[start_index * 9 + 0];
    double boussf_xy = boundary_vf[start_index * 9 + 1];
    double boussf_xz = boundary_vf[start_index * 9 + 2];
    double boussf_yx = boundary_vf[start_index * 9 + 3];
    double boussf_yy = boundary_vf[start_index * 9 + 4];
    double boussf_yz = boundary_vf[start_index * 9 + 5];
    double boussf_zx = boundary_vf[start_index * 9 + 6];
    double boussf_zy = boundary_vf[start_index * 9 + 7];
    double boussf_zz = boundary_vf[start_index * 9 + 8];
    int cellIndex = face2Cells[start_index];

    double bouDiv_x = bouSfx * boussf_xx + bouSfy * boussf_xy + bouSfz * boussf_xz;
    double bouDiv_y = bouSfx * boussf_yx + bouSfy * boussf_yy + bouSfz * boussf_yz;
    double bouDiv_z = bouSfx * boussf_zx + bouSfy * boussf_zy + bouSfz * boussf_zz;

    atomicAdd(&(output[cellIndex * 3 + 0]), bouDiv_x);
    atomicAdd(&(output[cellIndex * 3 + 1]), bouDiv_y);
    atomicAdd(&(output[cellIndex * 3 + 2]), bouDiv_z);
}

void permute_vector_d2h(hipStream_t stream, int num_cells, const double *input, double *output)
{
    size_t threads_per_block = 256;
    size_t blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    permute_vector_d2h_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, input, output);
}

void permute_vector_h2d(hipStream_t stream, int num_cells, const double *input, double *output)
{
    size_t threads_per_block = 256;
    size_t blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    permute_vector_h2d_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, input, output);
}

void field_multiply_scalar(hipStream_t stream,
        int num_cells, const double *input1, const double *input2, double *output,
        int num_boundary_surfaces, const double *boundary_input1, const double *boundary_input2, double *boundary_output)
{
    size_t threads_per_block = 256;
    size_t blocks_per_grid = (std::max(num_cells, num_boundary_surfaces) + threads_per_block - 1) / threads_per_block;
    field_multiply_scalar_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, num_boundary_surfaces,
            input1, input2, output, boundary_input1, boundary_input2, boundary_output);
}

void fvc_to_source_vector(hipStream_t stream, int num_cells, const double *volume, const double *fvc_output, double *source)
{
    size_t threads_per_block = 256;
    size_t blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    fvc_to_source_vector_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells,
            volume, fvc_output, source);
}

void ldu_to_csr(hipStream_t stream, int num_cells, int num_surfaces,
        const int *lower_to_csr_index, const int *upper_to_csr_index, const int *diag_to_csr_index,
        const double *lower, const double *upper, const double *diag, const double *source,
        const double *internal_coeffs, const double *boundary_coeffs,
        double *A, double *b)
{

}

void update_boundary_coeffs_vector(hipStream_t stream, int num_patches,
        const int *patch_size, const int *patch_type,
        double *value_internal_coeffs, double *value_boundary_coeffs,
        double *gradient_internal_coeffs, double *gradient_boundary_coeffs)
{
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = 1;

    int offset = 0;
    for (int i = 0; i < num_patches; i++) {
        threads_per_block = 256;
        blocks_per_grid = (patch_size[i] + threads_per_block - 1) / threads_per_block;
        // TODO: just basic patch type now
        // TODO: just vector version now
        if (patch_type[i] == boundaryConditions::zeroGradient) {
            update_boundary_coeffs_zeroGradient_vector<<<blocks_per_grid, threads_per_block, 0, stream>>>(patch_size[i], offset,
                    value_internal_coeffs, value_boundary_coeffs, gradient_internal_coeffs, gradient_boundary_coeffs);
        } else if (0) {
            // xxx
            fprintf(stderr, "boundaryConditions other than zeroGradient are not support yet!\n");
        }
        offset += patch_size[i];
    }
}

void fvm_ddt_vector(hipStream_t stream, int num_cells, double rDeltaT,
        const double *rho, const double *rho_old, const double *vf, const double *volume,
        double *diag, double *source, double sign)
{
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    fvm_ddt_vector_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells,
            rDeltaT, rho, rho_old, vf, volume, diag, source, sign);
}

void fvm_div_vector(hipStream_t stream, int num_surfaces, const int *lowerAddr, const int *upperAddr,
        const double *phi, const double *weight,
        double *lower, double *upper, double *diag, // end for internal
        int num_patches, const int *patch_size, const int *patch_type,
        const double *boundary_phi, const double *value_internal_coeffs, const double *value_boundary_coeffs,
        double *internal_coeffs, double *boundary_coeffs, double sign)
{
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = 1;

    blocks_per_grid = (num_surfaces + threads_per_block - 1) / threads_per_block;
    fvm_div_vector_internal<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_surfaces, lowerAddr, upperAddr,
            phi, weight, lower, upper, diag, sign);

    int offset = 0;
    for (int i = 0; i < num_patches; i++) {
        threads_per_block = 256;
        blocks_per_grid = (patch_size[i] + threads_per_block - 1) / threads_per_block;
        // TODO: just basic patch type now
        if (patch_type[i] == boundaryConditions::zeroGradient
                || patch_type[i] == boundaryConditions::fixedValue) {
            // TODO: just vector version now
            fvm_div_vector_boundary<<<blocks_per_grid, threads_per_block, 0, stream>>>(patch_size[i], offset,
                    boundary_phi, value_internal_coeffs, value_boundary_coeffs,
                    internal_coeffs, boundary_coeffs, sign);
        } else if (0) {
            // xxx
            fprintf(stderr, "boundaryConditions other than zeroGradient are not support yet!\n");
        }
        offset += patch_size[i];
    }
}

void fvm_laplacian_vector(hipStream_t stream, int num_surfaces,
        const int *lowerAddr, const int *upperAddr,
        const double *weight, const double *mag_sf, const double *delta_coeffs, const double *gamma,
        double *lower, double *upper, double *diag, // end for internal
        int num_patches, const int *patch_size, const int *patch_type,
        const double *boundary_mag_sf, const double *boundary_gamma,
        const double *gradient_internal_coeffs, const double *gradient_boundary_coeffs,
        double *internal_coeffs, double *boundary_coeffs, double sign)
{
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = 1;

    blocks_per_grid = (num_surfaces + threads_per_block - 1) / threads_per_block;
    fvm_laplacian_vector_internal<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_surfaces, lowerAddr, upperAddr,
            weight, mag_sf, delta_coeffs, gamma, lower, upper, diag, sign);

    int offset = 0;
    for (int i = 0; i < num_patches; i++) {
        threads_per_block = 256;
        blocks_per_grid = (patch_size[i] + threads_per_block - 1) / threads_per_block;
        // TODO: just basic patch type now
        if (patch_type[i] == boundaryConditions::zeroGradient
                || patch_type[i] == boundaryConditions::fixedValue) {
            // TODO: just vector version now
            fvm_laplacian_vector_boundary<<<blocks_per_grid, threads_per_block, 0, stream>>>(patch_size[i], offset,
                    boundary_mag_sf, boundary_gamma, gradient_internal_coeffs, gradient_boundary_coeffs,
                    internal_coeffs, boundary_coeffs, sign);
        } else if (0) {
            // xxx
            fprintf(stderr, "boundaryConditions other than zeroGradient are not support yet!\n");
        }
        offset += patch_size[i];
    }
}

void fvc_ddt_scalar(hipStream_t stream, int num_cells, double rDeltaT,
        const double *rho, const double *rho_old, const double *vf, const double *vf_old,
        double *output, double sign)
{
    checkCudaErrors(hipMemsetAsync(output, 0, num_cells * sizeof(double), stream));

    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    fvc_ddt_scalar_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells,
            rDeltaT, rho, rho_old, vf, vf_old, output, sign);
}

void fvc_grad_vector(hipStream_t stream, int num_cells, int num_surfaces, 
        const int *lowerAddr, const int *upperAddr, 
        const double *weight, const double *Sf, const double *vf, double *output, // end for internal
        int num_patches, const int *patch_size, const int *patch_type,
        const int *boundary_cell_face, const double *boundary_vf, const double *boundary_Sf,
        const double *volume, const double *boundary_mag_Sf, double *boundary_output,
        const double *boundary_deltaCoeffs, double sign)
{
    checkCudaErrors(hipMemsetAsync(output, 0, num_cells * 9 * sizeof(double), stream));
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (num_surfaces + threads_per_block - 1) / threads_per_block;
    fvc_grad_vector_internal<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_surfaces, lowerAddr, upperAddr,
            Sf, weight, vf, output);
    
    int offset = 0;
    // finish conctruct grad field except dividing cell volume
    for (int i = 0; i < num_patches; i++) {
        threads_per_block = 256;
        blocks_per_grid = (patch_size[i] + threads_per_block - 1) / threads_per_block;
        // TODO: just basic patch type now
        if (patch_type[i] == boundaryConditions::zeroGradient
                || patch_type[i] == boundaryConditions::fixedValue) {
            // TODO: just vector version now
            fvc_grad_vector_boundary<<<blocks_per_grid, threads_per_block, 0, stream>>>(patch_size[i], offset, boundary_cell_face,
                    boundary_Sf, boundary_vf, output);
        } else if (0) {
            // xxx
            fprintf(stderr, "boundaryConditions other than zeroGradient are not support yet!\n");
        }
        offset += patch_size[i];
    }

    // divide cell volume
    threads_per_block = 512;
    blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    divide_cell_volume_tsr<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, volume, output, sign);

    // correct boundary conditions
    offset = 0;
    for (int i = 0; i < num_patches; i++) {
        threads_per_block = 256;
        blocks_per_grid = (patch_size[i] + threads_per_block - 1) / threads_per_block;
        // TODO: just basic patch type now
        if (patch_type[i] == boundaryConditions::zeroGradient) {
            // TODO: just vector version now
            fvc_grad_vector_correctBC_zeroGradient<<<blocks_per_grid, threads_per_block, 0, stream>>>(patch_size[i], offset, boundary_cell_face,
                    output, boundary_vf, boundary_Sf, boundary_mag_Sf, boundary_output, sign);
        } else if (patch_type[i] == boundaryConditions::fixedValue) {
            fvc_grad_vector_correctBC_fixedValue<<<blocks_per_grid, threads_per_block, 0, stream>>>(patch_size[i], offset, boundary_cell_face,
                    output, boundary_vf, boundary_Sf, boundary_mag_Sf, boundary_output, boundary_deltaCoeffs, boundary_vf, sign);
        } else if (0) {
            // xxx
            fprintf(stderr, "boundaryConditions other than zeroGradient are not support yet!\n");
        }
        offset += patch_size[i];
    }
}

void scale_dev2T_tensor(hipStream_t stream, int num_cells, const double *vf1, double *vf2,
        int num_boundary_surfaces, const double *boundary_vf1, double *boundary_vf2)
{
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    scale_dev2t_tensor_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, vf1, vf2);

    blocks_per_grid = (num_boundary_surfaces + threads_per_block - 1) / threads_per_block;
    scale_dev2t_tensor_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_boundary_surfaces, boundary_vf1, boundary_vf2);
}

void fvc_div_surface_scalar(hipStream_t stream, int num_cells, int num_surfaces, int num_boundary_surfaces,
        const int *lowerAddr, const int *upperAddr, const double *ssf, const int *boundary_cell_face,
        const double *boundary_ssf, const double *volume, double *output, double sign)
{
    checkCudaErrors(hipMemsetAsync(output, 0, num_cells * sizeof(double), stream));

    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (num_surfaces + threads_per_block - 1) / threads_per_block;
    fvc_div_surface_scalar_internal<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_surfaces, lowerAddr, upperAddr, ssf, output);

    threads_per_block = 1024;
    blocks_per_grid = (num_boundary_surfaces + threads_per_block - 1) / threads_per_block;
    fvc_div_surface_scalar_boundary<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_boundary_surfaces, boundary_cell_face, 
            boundary_ssf, output);

    // divide cell volume
    threads_per_block = 1024;
    blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    divide_cell_volume_scalar<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, volume, output, sign);
}

void fvc_div_cell_vector(hipStream_t stream, int num_cells, int num_surfaces,
        const int *lowerAddr, const int *upperAddr, 
        const double *weight, const double *Sf, const double *vf, double *output, // end for internal
        int num_patches, const int *patch_size, const int *patch_type,
        const int *boundary_cell_face, const double *boundary_vf, const double *boundary_Sf,
        const double *volume, double sign)
{
    checkCudaErrors(hipMemsetAsync(output, 0, num_cells * sizeof(double), stream));

    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (num_surfaces + threads_per_block - 1) / threads_per_block;
    fvc_div_cell_vector_internal<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_surfaces, lowerAddr, upperAddr, vf, weight, Sf, output);

    int offset = 0;
    for (int i = 0; i < num_patches; i++) {
        threads_per_block = 256;
        blocks_per_grid = (patch_size[i] + threads_per_block - 1) / threads_per_block;
        // TODO: just basic patch type now
        if (patch_type[i] == boundaryConditions::zeroGradient
                || patch_type[i] == boundaryConditions::fixedValue) {
            // TODO: just vector version now
            fvc_div_cell_vector_boundary<<<blocks_per_grid, threads_per_block, 0, stream>>>(patch_size[i], offset, boundary_cell_face,
                    boundary_Sf, boundary_vf, output);
        } else if (0) {
            // xxx
            fprintf(stderr, "boundaryConditions other than zeroGradient are not support yet!\n");
        }
        offset += patch_size[i];
    }

    // divide cell volume
    threads_per_block = 1024;
    blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    divide_cell_volume_scalar<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, volume, output, sign);
}

void fvc_div_cell_tensor(hipStream_t stream, int num_cells, int num_surfaces,
        const int *lowerAddr, const int *upperAddr,
        const double *weight, const double *Sf, const double *vf, double *output, // end for internal
        int num_patches, const int *patch_size, const int *patch_type,
        const int *boundary_cell_face, const double *boundary_vf, const double *boundary_Sf,
        const double *volume, double sign)
{
    checkCudaErrors(hipMemsetAsync(output, 0, num_cells * 3 * sizeof(double), stream));

    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (num_surfaces + threads_per_block - 1) / threads_per_block;
    fvc_div_cell_tensor_internal<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_surfaces, lowerAddr, upperAddr, vf, weight, Sf, output);

    int offset = 0;
    for (int i = 0; i < num_patches; i++) {
        threads_per_block = 256;
        blocks_per_grid = (patch_size[i] + threads_per_block - 1) / threads_per_block;
        // TODO: just basic patch type now
        if (patch_type[i] == boundaryConditions::zeroGradient
                || patch_type[i] == boundaryConditions::fixedValue) {
            // TODO: just vector version now
            fvc_div_cell_tensor_boundary<<<blocks_per_grid, threads_per_block, 0, stream>>>(patch_size[i], offset, boundary_cell_face,
                    boundary_Sf, boundary_vf, output);
        } else if (0) {
            // xxx
            fprintf(stderr, "boundaryConditions other than zeroGradient are not support yet!\n");
        }
        offset += patch_size[i];
    }

    // divide cell volume
    threads_per_block = 1024;
    blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    divide_cell_volume_vec<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, volume, output, sign);
}

void fvc_grad_cell_scalar(hipStream_t stream, int num_cells, int num_surfaces, 
        const int *lowerAddr, const int *upperAddr, 
        const double *weight, const double *Sf, const double *vf, double *output, // end for internal
        int num_patches, const int *patch_size, const int *patch_type,
        const int *boundary_cell_face, const double *boundary_vf, const double *boundary_Sf, const double *volume, double sign)
{
    checkCudaErrors(hipMemsetAsync(output, 0, num_cells * 3 * sizeof(double), stream));
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (num_surfaces + threads_per_block - 1) / threads_per_block;
    fvc_grad_scalar_internal<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, num_surfaces, lowerAddr, upperAddr,
            Sf, weight, vf, output);
    
    int offset = 0;
    for (int i = 0; i < num_patches; i++) {
        threads_per_block = 256;
        blocks_per_grid = (patch_size[i] + threads_per_block - 1) / threads_per_block;
        // TODO: just non-coupled patch type now
        if (patch_type[i] == boundaryConditions::zeroGradient
                || patch_type[i] == boundaryConditions::fixedValue) {
            fvc_grad_scalar_boundary<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, patch_size[i], offset, boundary_cell_face,
                    boundary_Sf, boundary_vf, output);
        } else if (0) {
            // xxx
            fprintf(stderr, "boundaryConditions other than zeroGradient are not support yet!\n");
        }
        offset += patch_size[i];
    }

    // divide cell volume
    threads_per_block = 1024;
    blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    divide_cell_volume_vec<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, volume, output, sign);
}
